#include "jetstream/waterfall/cuda.hpp"

namespace Jetstream::Waterfall {

CUDA::CUDA(Config& c) : Generic(c) {
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    auto render = cfg.render;

    ymax = cfg.height;
    hipMalloc(&out_dptr, in.buf.size() * ymax * sizeof(float));

    binTextureCfg.buffer = (uint8_t*)out_dptr;
    binTextureCfg.cudaInterop = true;
    JETSTREAM_ASSERT_SUCCESS(this->_initRender());
}

CUDA::~CUDA() {
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::_compute() {
    hipMemcpyAsync(out_dptr+(inc*in.buf.size()), in.buf.data(), sizeof(float)*in.buf.size(),
            hipMemcpyDeviceToDevice, stream);
    hipStreamSynchronize(stream);

    return Result::SUCCESS;
}

Result CUDA::_present() {
    binTexture->fill();

    return Result::SUCCESS;
}

} // namespace Jetstream::Waterfall
