#include "hip/hip_runtime.h"
#include "jetstream/fft/cuda.hpp"

namespace Jetstream::FFT {

static __device__ inline float clamp(float x, float a, float b) {
    return max(a, min(b, x));
}

static __device__ inline float scale(float x, float min, float max) {
    return (x - min) / (max - min);
}

static __device__ inline float amplt(hipFloatComplex x, int n) {
    return 20 * log10(hipCabsf(x) / n);
}

static __device__ inline int shift(int i, uint n) {
    return (i + (n / 2) - 1) % n;
}

static __global__ void pre(hipfftComplex* c, const hipfftComplex* win, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < n; i += numThreads) {
        c[i] = hipCmulf(c[i], win[i]);
    }
}

static __global__ void post(const hipfftComplex* c, float* r,
      const float min, const float max, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < n; i += numThreads) {
        r[i] = amplt(c[shift(i, n)], n);
        r[i] = scale(r[i], min, max);
        r[i] = clamp(r[i], 0.0, 1.0);
    }
}

CUDA::CUDA(Config& c) : Generic(c) {
    fft_len = in.buf.size() * sizeof(float) * 2;
    hipMalloc(&fft_dptr, fft_len);

    win_len = in.buf.size() * sizeof(float) * 2;
    hipMalloc(&win_dptr, win_len);
    hipMemcpy(win_dptr, window.data(), win_len, hipMemcpyHostToDevice);

    out_len = in.buf.size() * sizeof(float);
    hipMallocManaged(&out_dptr, out_len);
    out.buf = nonstd::span<float>{out_dptr, in.buf.size()};

    hipMemAdvise(out_dptr, out_len, hipMemAdviseSetPreferredLocation, 0);
    hipMemAdvise(out_dptr, out_len, hipMemAdviseSetReadMostly, hipCpuDeviceId);

    hipfftPlan1d(&plan, in.buf.size(), HIPFFT_C2C, 1);
}

CUDA::~CUDA() {
    hipfftDestroy(plan);
    hipFree(fft_dptr);
    hipFree(out_dptr);
}

Result CUDA::underlyingCompute() {
    hipMemcpy(fft_dptr, in.buf.data(), fft_len, hipMemcpyHostToDevice);
    pre<<<4, 512>>>(fft_dptr, win_dptr, in.buf.size());
    hipfftExecC2C(plan, fft_dptr, fft_dptr, HIPFFT_FORWARD);
    post<<<4, 512>>>(fft_dptr, out_dptr, cfg.min_db, cfg.max_db, in.buf.size());
    hipDeviceSynchronize();
    hipMemPrefetchAsync(out_dptr, out_len, hipCpuDeviceId);

    return Result::SUCCESS;
}

Result CUDA::underlyingPresent() {
    return Result::SUCCESS;
}

} // namespace Jetstream::FFT
