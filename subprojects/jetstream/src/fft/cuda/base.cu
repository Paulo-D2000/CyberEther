#include "hip/hip_runtime.h"
#include "jetstream/fft/cuda.hpp"

namespace Jetstream::FFT {

__device__ inline float clamp(float x, float a, float b) {
  return max(a, min(b, x));
}

__device__ inline float scale(float x, float min, float max) {
  return (x - min) / (max - min);
}

__device__ inline float amplt(hipFloatComplex x, int n) {
  return 20 * log10(hipCabsf(x) / n);
}

__device__ inline int shift(int i, uint n) {
  return (i + (n / 2) - 1) % n;
}

__global__ void post_process(const hipfftComplex* c, float* r,
    const float min, const float max, const uint n){
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
      i += blockDim.x * gridDim.x) {
    r[i] = amplt(c[shift(i, n)], n);
    r[i] = scale(r[i], min, max);
    r[i] = clamp(r[i], 0.0, 1.0);
  }
}

CUDA::CUDA(Config& c) : Generic(c) {
  fft_len = in.buf.size() * sizeof(float) * 2;
  hipMalloc(&fft_dptr, fft_len);

  out_len = in.buf.size() * sizeof(float);
  hipMallocManaged(&out_dptr, out_len);
  out.buf = nonstd::span<float>{out_dptr, in.buf.size()};

  hipMemAdvise(out_dptr, out_len, hipMemAdviseSetPreferredLocation, 0);
  hipMemAdvise(out_dptr, out_len, hipMemAdviseSetReadMostly, hipCpuDeviceId);

  hipfftPlan1d(&plan, in.buf.size(), HIPFFT_C2C, 1);
}

CUDA::~CUDA() {
  hipfftDestroy(plan);
  hipFree(fft_dptr);
  hipFree(out_dptr);
}

Result CUDA::underlyingCompute() {
  hipMemcpy(fft_dptr, in.buf.data(), fft_len, hipMemcpyHostToDevice);
  hipfftExecC2C(plan, fft_dptr, fft_dptr, HIPFFT_FORWARD);
  post_process<<<CB(in.buf.size()), kNumBlockThreads>>>
    (fft_dptr, out_dptr, cfg.min_db, cfg.max_db, in.buf.size());
  hipDeviceSynchronize();
  hipMemPrefetchAsync(out_dptr, out_len, hipCpuDeviceId);

  return Result::SUCCESS;
}

Result CUDA::underlyingPresent() {
  return Result::SUCCESS;
}

} // namespace Jetstream::FFT
