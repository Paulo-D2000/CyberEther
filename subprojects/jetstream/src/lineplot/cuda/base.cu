#include "jetstream/lineplot/cuda.hpp"

namespace Jetstream::Lineplot {

CUDA::CUDA(Config& c) : Generic(c) {
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    plot_len = plot.size() * sizeof(plot[0]);
    CUDA_CHECK_THROW(hipMalloc(&plot_dptr, plot_len));
    CUDA_CHECK_THROW(hipMemcpy(plot_dptr, plot.data(), plot_len, hipMemcpyHostToDevice));

    plotVbo.data = plot_dptr;
    plotVbo.cudaInterop = true;
    JETSTREAM_CHECK_THROW(this->_initRender());
}

CUDA::~CUDA() {
    CUDA_CHECK_THROW(hipFree(plot_dptr));
    CUDA_CHECK_THROW(hipStreamDestroy(stream));
}

Result CUDA::_compute() {
    size_t elementSize = sizeof(float);
    size_t srcPitchInBytes = 1 * elementSize;
    size_t dstPitchInBytes = 3 * elementSize;
    size_t width = 1 * elementSize;
    size_t height = in.buf.size();

    CUDA_CHECK(hipMemcpy2DAsync(plot_dptr + 1, dstPitchInBytes, in.buf.data(), srcPitchInBytes,
        width, height, hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    return Result::SUCCESS;
}

Result CUDA::_present() {
    lineVertex->update();

    return Result::SUCCESS;
}

} // namespace Jetstream::Lineplot
