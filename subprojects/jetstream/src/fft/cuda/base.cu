#include "hip/hip_runtime.h"
#include "jetstream/fft/cuda.hpp"

namespace Jetstream::FFT {

static __device__ inline float clamp(float x, float a, float b) {
    return max(a, min(b, x));
}

static __device__ inline float scale(float x, float min, float max) {
    return (x - min) / (max - min);
}

static __device__ inline float amplt(hipFloatComplex x, int n) {
    return 20 * log10(hipCabsf(x) / n);
}

static __device__ inline int shift(int i, uint n) {
    return (i + (n / 2) - 1) % n;
}

static __global__ void pre(hipfftComplex* c, const hipfftComplex* win, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        c[i] = hipCmulf(c[i], win[i]);
    }
}

static __global__ void post(const hipfftComplex* c, float* r,
    const float min, const float max, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        float tmp;

        tmp = amplt(c[shift(i, n)], n);
        tmp = scale(tmp, min, max);
        tmp = clamp(tmp, 0.0, 1.0);

        r[i] = tmp;
    }
}

CUDA::CUDA(Config& c) : Generic(c) {
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK_THROW(hipHostRegister(in.buf.data(), in.buf.size() * sizeof(in.buf[0]),
            hipHostRegisterReadOnly));

    fft_len = in.buf.size() * sizeof(in.buf[0]);
    CUDA_CHECK_THROW(hipMalloc(&fft_dptr, fft_len));

    win_len = in.buf.size() * sizeof(in.buf[0]);
    CUDA_CHECK_THROW(hipMalloc(&win_dptr, win_len));
    CUDA_CHECK_THROW(hipMemcpy(win_dptr, window.data(), win_len, hipMemcpyHostToDevice));

    out_len = in.buf.size() * sizeof(float);
    CUDA_CHECK_THROW(hipMallocManaged(&out_dptr, out_len));
    out.buf = nonstd::span<float>{out_dptr, in.buf.size()};

    CUDA_CHECK_THROW(hipfftPlan1d(&plan, in.buf.size(), HIPFFT_C2C, 1));
    CUDA_CHECK_THROW(hipfftSetStream(plan, stream));
}

CUDA::~CUDA() {
    CUDA_CHECK_THROW(hipHostUnregister(in.buf.data()));
    CUDA_CHECK_THROW(hipfftDestroy(plan));
    CUDA_CHECK_THROW(hipFree(fft_dptr));
    CUDA_CHECK_THROW(hipFree(out_dptr));
    CUDA_CHECK_THROW(hipStreamDestroy(stream));
}

Result CUDA::underlyingCompute() {
    int N = in.buf.size();
    int threads = 32;
    int blocks = (N + threads - 1) / threads;

    CUDA_CHECK(hipMemcpyAsync(fft_dptr, in.buf.data(), fft_len, hipMemcpyHostToDevice, stream));
    pre<<<blocks, threads, 0, stream>>>(fft_dptr, win_dptr, N);
    CUDA_CHECK(hipfftExecC2C(plan, fft_dptr, fft_dptr, HIPFFT_FORWARD));
    post<<<blocks, threads, 0, stream>>>(fft_dptr, out_dptr, cfg.min_db, cfg.max_db, N);
    CUDA_CHECK(hipStreamSynchronize(stream));

    return Result::SUCCESS;
}

Result CUDA::underlyingPresent() {
    return Result::SUCCESS;
}

} // namespace Jetstream::FFT
