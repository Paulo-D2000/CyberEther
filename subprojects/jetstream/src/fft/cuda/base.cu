#include "hip/hip_runtime.h"
#include "jetstream/fft/cuda.hpp"

namespace Jetstream::FFT {

static __device__ inline float clamp(float x, float a, float b) {
    return max(a, min(b, x));
}

static __device__ inline float scale(float x, float min, float max) {
    return (x - min) / (max - min);
}

static __device__ inline float amplt(hipFloatComplex x, int n) {
    return 20 * log10(hipCabsf(x) / n);
}

static __device__ inline int shift(int i, uint n) {
    return (i + (n / 2) - 1) % n;
}

static __global__ void pre(hipfftComplex* c, const hipfftComplex* win, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        c[i] = hipCmulf(c[i], win[i]);
    }
}

static __global__ void post(const hipfftComplex* c, float* r,
      const float min, const float max, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        float tmp;

        tmp = amplt(c[shift(i, n)], n);
        tmp = scale(tmp, min, max);
        tmp = clamp(tmp, 0.0, 1.0);

        r[i] = tmp;
    }
}

CUDA::CUDA(Config& c) : Generic(c) {
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    fft_len = in.buf.size() * sizeof(float) * 2;
    hipMalloc(&fft_dptr, fft_len);

    win_len = in.buf.size() * sizeof(float) * 2;
    hipMalloc(&win_dptr, win_len);
    hipMemcpy(win_dptr, window.data(), win_len, hipMemcpyHostToDevice);

    out_len = in.buf.size() * sizeof(float);
    hipMallocManaged(&out_dptr, out_len);
    out.buf = nonstd::span<float>{out_dptr, in.buf.size()};

    hipfftPlan1d(&plan, in.buf.size(), HIPFFT_C2C, 1);
    hipfftSetStream(plan, stream);
}

CUDA::~CUDA() {
    hipfftDestroy(plan);
    hipFree(fft_dptr);
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::underlyingCompute() {
    int N = in.buf.size();
    int threads = 32;
    int blocks = (N + threads - 1) / threads;

    hipMemcpyAsync(fft_dptr, in.buf.data(), fft_len, hipMemcpyHostToDevice, stream);
    pre<<<blocks, threads, 0, stream>>>(fft_dptr, win_dptr, N);
    hipfftExecC2C(plan, fft_dptr, fft_dptr, HIPFFT_FORWARD);
    post<<<blocks, threads, 0, stream>>>(fft_dptr, out_dptr, cfg.min_db, cfg.max_db, N);
    hipStreamSynchronize(stream);

    return Result::SUCCESS;
}

Result CUDA::underlyingPresent() {
    return Result::SUCCESS;
}

} // namespace Jetstream::FFT
