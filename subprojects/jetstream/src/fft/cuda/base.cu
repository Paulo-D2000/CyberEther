#include "hip/hip_runtime.h"
#include "jetstream/fft/cuda.hpp"

namespace Jetstream::FFT {

static __device__ inline float clamp(float x, float a, float b) {
    return max(a, min(b, x));
}

static __device__ inline float scale(float x, float min, float max) {
    return (x - min) / (max - min);
}

static __device__ inline float amplt(hipFloatComplex x, int n) {
    return 20 * log10(hipCabsf(x) / n);
}

static __device__ inline int shift(int i, uint n) {
    return (i + (n / 2) - 1) % n;
}

static __global__ void pre(hipfftComplex* c, const hipfftComplex* win, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        c[i] = hipCmulf(c[i], win[i]);
    }
}

static __global__ void post(const hipfftComplex* c, float* r,
    const float min, const float max, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        float tmp;

        tmp = amplt(c[shift(i, n)], n);
        tmp = scale(tmp, min, max);
        tmp = clamp(tmp, 0.0, 1.0);

        r[i] = tmp;
    }
}

CUDA::CUDA(Config& c) : Generic(c) {
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK_THROW(hipHostRegister(in.buf.data(), in.buf.size() * sizeof(in.buf[0]),
            hipHostRegisterReadOnly));

    fft_len = in.buf.size() * sizeof(in.buf[0]);
    CUDA_CHECK_THROW(hipMalloc(&fft_dptr, fft_len));

    win_len = in.buf.size() * sizeof(in.buf[0]);
    CUDA_CHECK_THROW(hipMalloc(&win_dptr, win_len));
    CUDA_CHECK_THROW(hipMemcpy(win_dptr, window.data(), win_len, hipMemcpyHostToDevice));

    out_len = in.buf.size() * sizeof(float);
    CUDA_CHECK_THROW(hipMallocManaged(&out_dptr, out_len));
    out.buf = nonstd::span<float>{out_dptr, in.buf.size()};

    hipfftPlan1d(&plan, in.buf.size(), HIPFFT_C2C, 1);
    hipfftSetStream(plan, stream);
}

CUDA::~CUDA() {
    hipHostUnregister(in.buf.data());
    hipfftDestroy(plan);
    hipFree(fft_dptr);
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::underlyingCompute() {
    DEBUG_PUSH("fft_compute");

    int N = in.buf.size();
    int threads = 32;
    int blocks = (N + threads - 1) / threads;

    CUDA_CHECK(hipMemcpyAsync(fft_dptr, in.buf.data(), fft_len, hipMemcpyHostToDevice, stream));
    pre<<<blocks, threads, 0, stream>>>(fft_dptr, win_dptr, N);
    hipfftExecC2C(plan, fft_dptr, fft_dptr, HIPFFT_FORWARD);
    post<<<blocks, threads, 0, stream>>>(fft_dptr, out_dptr, cfg.min_db, cfg.max_db, N);
    CUDA_CHECK(hipStreamSynchronize(stream));

    DEBUG_POP();
    return Result::SUCCESS;
}

Result CUDA::underlyingPresent() {
    return Result::SUCCESS;
}

} // namespace Jetstream::FFT
