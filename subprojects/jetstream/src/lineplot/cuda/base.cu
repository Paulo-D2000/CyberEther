#include "jetstream/lineplot/cuda.hpp"

namespace Jetstream::Lineplot {

CUDA::CUDA(Config& c) : Generic(c) {
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    plot_len = plot.size() * sizeof(plot[0]);
    hipMalloc(&plot_dptr, plot_len);
    hipMemcpy(plot_dptr, plot.data(), plot_len, hipMemcpyHostToDevice);

    plotVbo.data = plot_dptr;
    plotVbo.cudaInterop = true;
    JETSTREAM_ASSERT_SUCCESS(this->_initRender());
}

CUDA::~CUDA() {
    hipFree(plot_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::_compute() {
    size_t elementSize = sizeof(float);
    size_t srcPitchInBytes = 1 * elementSize;
    size_t dstPitchInBytes = 3 * elementSize;
    size_t width = 1 * elementSize;
    size_t height = in.buf.size();

    hipMemcpy2DAsync(plot_dptr + 1, dstPitchInBytes, in.buf.data(), srcPitchInBytes,
        width, height, hipMemcpyDeviceToDevice, stream);
    hipStreamSynchronize(stream);

    return Result::SUCCESS;
}

Result CUDA::_present() {
    lineVertex->update();

    return Result::SUCCESS;
}

} // namespace Jetstream::Lineplot
