#include "jetstream/waterfall/cuda.hpp"

namespace Jetstream::Waterfall {

CUDA::CUDA(Config& c) : Generic(c) {
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    auto render = cfg.render;

    ymax = cfg.height;
    CUDA_CHECK_THROW(hipMalloc(&out_dptr, in.buf.size() * ymax * sizeof(float)));

    binTextureCfg.buffer = (uint8_t*)out_dptr;
    binTextureCfg.cudaInterop = true;
    JETSTREAM_CHECK_THROW(this->_initRender());
}

CUDA::~CUDA() {
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::_compute() {
    CUDA_CHECK(hipMemcpyAsync(out_dptr+(inc*in.buf.size()), in.buf.data(), sizeof(float)*in.buf.size(),
            hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    return Result::SUCCESS;
}

} // namespace Jetstream::Waterfall
